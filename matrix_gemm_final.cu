//header files included
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <string>
#include <sstream>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <ctime>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

//declaring the tile width and height
//for tile based matrix multiplication
#define TILE_WIDTH 64
#define TILE_HEIGHT 64

//Namespace for std
using namespace std;

//structure declaration for storing rows and columns for a matrix
struct matrix{
    unsigned int rows;	//storing rows of a matrix
    unsigned int cols;	//storing columns of a matrix
};








//handle error alias name declaration
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

//global kernal for matrix multiplication, takes in input matrices and sizes, and multiplies them
//matrix multiplication is being done tile by tile
__global__ void matrix_mult(float* array1, unsigned int rows1, unsigned int cols1, float* array2, unsigned int rows2, unsigned int cols2, float* array3)
{
    //shared memory takes one tile at a time
    __shared__ float S1[TILE_WIDTH][TILE_HEIGHT];	//to store tiles for array 1
    __shared__ float S2[TILE_HEIGHT][TILE_WIDTH];	//to store tiles for array 2

    //threads x and y index for the current block
    unsigned int tx=threadIdx.x;
    unsigned int ty=threadIdx.y;

    unsigned int c=blockIdx.x*blockDim.x + threadIdx.x;	//row value using x-index of current thread
    unsigned int r=blockIdx.y*blockDim.y + threadIdx.y;	//column value using y-index of current thread

    unsigned int idx=c*rows1+r;				//column major index, using row and column value

    float val=0;		//register to store multiplication result initialized to zero

    for(int m=0; m<1+((rows2-1)/TILE_WIDTH);m++)	//going over all tiles one by one, with each m
    {

        int var1=m*TILE_WIDTH+tx ;		//x thread value for current tile
        int var2=m*TILE_WIDTH+ty ;		//y thread value for current tile

        //copying a tile from array1
        if (r < rows1 && var1 < rows2)		//if the value is associated to a valid matrix coordinate in array1 then store it to shared memory S1
            S1[ty][tx]=array1[r + var1*rows1];//storing a "valid" value from array to shared memory
        else
            S1[ty][tx]=0;					//storing zero, since there is no valid value
        __syncthreads();						//syncing all threads once shared memory S1 is stored

        //copying a tile from array2
        if(c < cols2 && var2 < rows2)	//if value is associates to a valid matrix coordinate in array2 then store it to shared memory S2
            S2[ty][tx]=array2[var2+rows2*c];	//storing the valid value
        else
            S2[ty][tx]=0;		//storing zero, since no valid value
        __syncthreads();		//synchronizing threads


        for(int i=0; i<TILE_WIDTH;i++)	//going over entire tile, ty row in S1 and tx column in S2
            val+=S1[ty][i]*S2[i][tx];	//and multiplying elements
        __syncthreads();		//synchronizing threads

    }

    if(r < rows1 && c< cols2)	//removing degenerate cases
        array3[idx]=val;	//saving multiplication result to global memory

}

int main(int argc, char* argv[])
{
    if(argc != 4) //there should be four arguments, Usage: prog matrix1.mtx matrix2.mtx matrix3.mtx
        return 1; //exit and return an error

    ifstream infile_A, infile_B;	//reading the input matrices

    // *****************************************************************************
    //                                   Matrix A                                 //
    //******************************************************************************

    infile_A.open(argv[1],ios::binary|ios::in|ios::ate);

    //getting end and beginning of the file
    infile_A.seekg(0,ios::end);
    infile_A.seekg(0,ios::beg);

    //memory allocation
    matrix M_A;
    infile_A.read(reinterpret_cast<char*>(&M_A),2*sizeof(unsigned int));


    float* array_A=(float*)malloc(M_A.rows*M_A.cols*sizeof(float));	//column major
    infile_A.read(reinterpret_cast<char*>(array_A),M_A.rows*M_A.cols);

    infile_A.close();
    

    // *****************************************************************************
    //                                   Matrix B                                 //
    //******************************************************************************

    infile_B.open(argv[2],ios::binary|ios::in|ios::ate);

    //getting end and beginning of the file
    infile_B.seekg(0,ios::end);
    infile_B.seekg(0,ios::beg);

    //memory allocation
    matrix M_B;
    infile_B.read(reinterpret_cast<char*>(&M_B),2*sizeof(unsigned int));


    float* array_B=(float*)malloc(M_B.rows*M_B.cols*sizeof(float));	//column major
    infile_B.read(reinterpret_cast<char*>(array_B),M_B.rows*M_B.cols);

    infile_B.close();

    if(M_A.cols!=M_B.rows) //checking if the two matrices can be multiplied
    {
        cout<<"Illegal matrix sizes: "<<M_A.cols<<" != "<<M_B.rows<<endl;
        return 1;
    }
    // *****************************************************************************
    //                                   allocate to the host                      //
    //******************************************************************************
    float* array_C=(float*)malloc(M_A.rows*M_B.cols*sizeof(float));//array to store gpu result in column major format
    //GPU DEVICE PROPERTIES and selecting a GPU for calculation
    int nDevices;
    hipGetDeviceCount(&nDevices);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);	//using GPU0

    //BLOCK AND GRID SIZE DECLARATION
    float thread_block=sqrt(prop.maxThreadsPerBlock);	//2D blocks used
    dim3 DimGrid(ceil(M_B.cols/thread_block),ceil(M_A.rows/thread_block),1); //image saved as a 2D grid
    dim3 DimBlock(thread_block,thread_block,1);

    size_t Sbytes = 2* DimBlock.x * DimBlock.y ;	//2 arrays used in the calculation, hence 2 * DimBlock.x * DimBlock.y

    //Checking if sufficient shared memory available or not

    if(prop.sharedMemPerBlock < Sbytes){
        std::cout<<"ERROR: insufficient shared memory"<<std::endl;
        exit(1);
    }

    // *****************************************************************************
    //                                   allocate to the GPU                      //
    //******************************************************************************

    float *array_A_gpu, *array_B_gpu, *array_C_gpu;//gpu arrays declared

    hipMalloc(&array_A_gpu,M_A.rows*M_A.cols*sizeof(float)); //allocate space to store arrayA

    hipMalloc(&array_B_gpu,M_B.rows*M_B.cols*sizeof(float)); //allocate space to store arrayB

    hipMalloc(&array_C_gpu,M_A.rows*M_B.cols*sizeof(float)); //allocate space to store gpu result

    //COPY TO GPU MEMORY
    hipMemcpy(array_A_gpu, array_A, M_A.rows*M_A.cols*sizeof(float), hipMemcpyHostToDevice);//copy arrayA to gpu

    hipMemcpy(array_B_gpu, array_B, M_B.rows*M_B.cols*sizeof(float), hipMemcpyHostToDevice);//copy arrayB to gpu

    hipMemcpy(array_C_gpu, array_C, M_A.rows*M_B.cols*sizeof(float), hipMemcpyHostToDevice);//copy arrayC to gpu

    // *****************************************************************************
    //                                   allocate to the GPU                      //
    //******************************************************************************

    //time measurement for matrix multiplication
    hipEvent_t start1, stop1;

    hipEventCreate(&start1);
    hipEventCreate(&stop1);

    //MATRIX MULTIPLICATION USING KERNEL
    hipEventRecord(start1);
    matrix_mult<<<DimGrid, DimBlock, Sbytes>>>(array_A_gpu,M_A.rows,M_A.cols,array_B_gpu,M_B.rows,M_B.cols,array_C_gpu);//calling the kernel
    hipEventRecord(stop1);

    hipEventSynchronize(stop1);

    float milliseconds1 = 0;//storing the execution time in milliseconds

    hipEventElapsedTime(&milliseconds1, start1, stop1);//get the time in milliseconds
    float msecPerMatrixMul = milliseconds1;
    double flopsPerMatrixMul = 2.0 * (double) M_A.rows *(double) M_B.cols *(double) M_A.cols;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
    printf("Performance= %.2f GFlop/s, Time= %.3f msec",gigaFlops,msecPerMatrixMul);

    //copy to CPU MEMORY
    hipMemcpy(array_C, array_C_gpu, M_A.rows*M_B.cols*sizeof(float), hipMemcpyDeviceToHost);//copying result of multiplication from gpu to cpu


    // *****************************************************************************
    //                                   Saving the result                        //
    //******************************************************************************
    //SAVING THE OUTPUT MATRIX
    ofstream ofile(argv[3], ios::binary);

    ofile.write((char*) &M_A.rows, sizeof(unsigned int));//writing the rows
    ofile.write((char*) &M_B.cols, sizeof(unsigned int));//writing the cols
    ofile.write((char*) array_C , M_A.rows*M_B.cols*sizeof(float));//writing all elements


    return 0;
}
