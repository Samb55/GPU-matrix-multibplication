//header files included
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <string>
#include <sstream>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <ctime>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

//declaring the tile width and height
//for tile based matrix multiplication
#define TILE_WIDTH 32
#define TILE_HEIGHT 32

//Namespace for std
using namespace std;

//structure declaration for storing rows and columns for a matrix
struct matrix{
    unsigned int rows;	//storing rows of a matrix
    unsigned int cols;	//storing columns of a matrix
};

//handlerror declaration : to display file and line numbers of erroneous lines
static void HandleError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        cout<<hipGetErrorString(err)<<" in "<< file <<" at line "<< line<<endl;
    }
}

//handle error alias name declaration
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

//global kernal for matrix multiplication, takes in input matrices and sizes, and multiplies them
//matrix multiplication is being done tile by tile
__global__ void matrix_mult(float* array1, unsigned int rows1, unsigned int cols1, float* array2, unsigned int rows2, unsigned int cols2, float* array3)
{
    //shared memory takes one tile at a time
    __shared__ float S1[TILE_WIDTH][TILE_HEIGHT];	//to store tiles for array 1
    __shared__ float S2[TILE_HEIGHT][TILE_WIDTH];	//to store tiles for array 2

    //threads x and y index for the current block
    unsigned int tx=threadIdx.x;
    unsigned int ty=threadIdx.y;

    unsigned int c=blockIdx.x*blockDim.x + threadIdx.x;	//row value using x-index of current thread
    unsigned int r=blockIdx.y*blockDim.y + threadIdx.y;	//column value using y-index of current thread

    unsigned int idx=c*rows1+r;				//column major index, using row and column value

    float val=0;		//register to store multiplication result initialized to zero

    for(int m=0; m<1+((rows2-1)/TILE_WIDTH);m++)	//going over all tiles one by one, with each m
    {

        int var1=m*TILE_WIDTH+tx ;		//x thread value for current tile
        int var2=m*TILE_WIDTH+ty ;		//y thread value for current tile

        //copying a tile from array1
        if (r < rows1 && var1 < rows2)		//if the value is associated to a valid matrix coordinate in array1 then store it to shared memory S1
            S1[ty][tx]=array1[r + var1*rows1];//storing a "valid" value from array to shared memory
        else
            S1[ty][tx]=0;					//storing zero, since there is no valid value
        __syncthreads();						//syncing all threads once shared memory S1 is stored

        //copying a tile from array2
        if(c < cols2 && var2 < rows2)	//if value is associates to a valid matrix coordinate in array2 then store it to shared memory S2
            S2[ty][tx]=array2[var2+rows2*c];	//storing the valid value
        else
            S2[ty][tx]=0;		//storing zero, since no valid value
        __syncthreads();		//synchronizing threads


        for(int i=0; i<TILE_WIDTH;i++)	//going over entire tile, ty row in S1 and tx column in S2
            val+=S1[ty][i]*S2[i][tx];	//and multiplying elements
        __syncthreads();		//synchronizing threads

    }

    if(r < rows1 && c< cols2)	//removing degenerate cases
        array3[idx]=val;	//saving multiplication result to global memory

}

int main(int argc, char* argv[])
{
    if(argc != 4) //there should be four arguments, Usage: prog matrix1.mtx matrix2.mtx matrix3.mtx
        return 1; //exit and return an error

    ifstream infile_A, infile_B;	//reading the input matrices

    // *****************************************************************************
    //                                   Matrix A                                 //
    //******************************************************************************

    infile_A.open(argv[1],ios::binary|ios::in|ios::ate);

    //getting end and beginning of the file
    infile_A.seekg(0,ios::end);
    infile_A.seekg(0,ios::beg);

    //memory allocation
    matrix M_A;
    infile_A.read(reinterpret_cast<char*>(&M_A),2*sizeof(unsigned int));


    float* array_A=(float*)malloc(M_A.rows*M_A.cols*sizeof(float));	//column major
    infile_A.read(reinterpret_cast<char*>(array_A),M_A.rows*M_A.cols);

    infile_A.close();
    

    // *****************************************************************************
    //                                   Matrix B                                 //
    //******************************************************************************

    infile_B.open(argv[2],ios::binary|ios::in|ios::ate);

    //getting end and beginning of the file
    infile_B.seekg(0,ios::end);
    infile_B.seekg(0,ios::beg);

    //memory allocation
    matrix M_B;
    infile_B.read(reinterpret_cast<char*>(&M_B),2*sizeof(unsigned int));


    float* array_B=(float*)malloc(M_B.rows*M_B.cols*sizeof(float));	//column major
    infile_B.read(reinterpret_cast<char*>(array_B),M_B.rows*M_B.cols);
   
    infile_B.close();

    if(M_A.cols!=M_B.rows) //checking if the two matrices can be multiplied
    {
        cout<<"Illegal matrix sizes: "<<M_A.cols<<" != "<<M_B.rows<<endl;
        return 1;
    }

    // *****************************************************************************
    //                                   allocate to the host                      //
    //******************************************************************************


    float* array_D=(float*)malloc(M_A.rows*M_B.cols*sizeof(float));//arary to store cublas result in column major format
    int nDevices;
    HANDLE_ERROR(hipGetDeviceCount(&nDevices));

    hipDeviceProp_t prop;
    HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));	//using GPU0

    //BLOCK AND GRID SIZE DECLARATION
    float thread_block=sqrt(prop.maxThreadsPerBlock);	//2D blocks used
    dim3 DimGrid(ceil(M_B.cols/thread_block),ceil(M_A.rows/thread_block),1); //image saved as a 2D grid
    dim3 DimBlock(thread_block,thread_block,1);

    size_t Sbytes = 2* DimBlock.x * DimBlock.y ;	//2 arrays used in the calculation, hence 2 * DimBlock.x * DimBlock.y

    //Checking if sufficient shared memory available or not

    if(prop.sharedMemPerBlock < Sbytes){
        std::cout<<"ERROR: insufficient shared memory"<<std::endl;
        exit(1);
    }

    // *****************************************************************************
    //                                   allocate to the GPU                      //
    //******************************************************************************

    float *array_A_gpu, *array_B_gpu,  *array_D_gpu;	//gpu arrays declared

    HANDLE_ERROR(hipMalloc(&array_A_gpu,M_A.rows*M_A.cols*sizeof(float))); //allocate space to store arrayA

    HANDLE_ERROR(hipMalloc(&array_B_gpu,M_B.rows*M_B.cols*sizeof(float))); //allocate space to store arrayB

    HANDLE_ERROR(hipMalloc(&array_D_gpu,M_A.rows*M_B.cols*sizeof(float))); //allocate space to store cublas result


    //COPY TO GPU MEMORY
    HANDLE_ERROR(hipMemcpy(array_A_gpu, array_A, M_A.rows*M_A.cols*sizeof(float), hipMemcpyHostToDevice));//copy arrayA to gpu

    HANDLE_ERROR(hipMemcpy(array_B_gpu, array_B, M_B.rows*M_B.cols*sizeof(float), hipMemcpyHostToDevice));//copy arrayB to gpu

    HANDLE_ERROR(hipMemcpy(array_D_gpu, array_D, M_A.rows*M_B.cols*sizeof(float), hipMemcpyHostToDevice));//copy arrayD to gpu

    // *****************************************************************************
    //                                   allocate to the GPU                      //
    //******************************************************************************

//Creating handle for CUBLAS
    float milliseconds2 = 0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    //parameter declaration for cublas implementation
    float alpha = 1.0;
    float beta = 0.0;

    //cublas time measurement
    hipEvent_t start2, stop2;

    hipEventCreate(&start2);
    hipEventCreate(&stop2);

    //MATRIX MULTIPLICATION USING CUBLAS

    hipEventRecord(start2);
    for( int s=0; s<20;s++){
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M_A.rows, M_B.cols, M_A.cols, &alpha, array_A_gpu, M_A.rows, array_B_gpu, M_B.rows, &beta, array_D_gpu, M_A.rows);
    }
    hipEventRecord(stop2);
    hipEventSynchronize(stop2);
    hipEventElapsedTime(&milliseconds2, start2, stop2);//get the time in milliseconds
    float msecPerMatrixMul = milliseconds2 / 20;
    double flopsPerMatrixMul = 2.0 * (double) M_A.rows *(double) M_B.cols *(double) M_A.cols;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
    printf("Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops\n",gigaFlops,msecPerMatrixMul, flopsPerMatrixMul);

    //copy to CPU MEMORY
    HANDLE_ERROR(hipMemcpy(array_D, array_D_gpu, M_A.rows*M_B.cols*sizeof(float), hipMemcpyDeviceToHost));//copy result of multiplication using CUBLAS from gpu to cpu



    // *****************************************************************************
    //                                   Saving the result                        //
    //******************************************************************************

    ofstream ofile(argv[3], ios::binary);

    ofile.write((char*) &M_A.rows, sizeof(unsigned int));
    ofile.write((char*) &M_B.cols, sizeof(unsigned int));
    ofile.write((char*) array_D , M_A.rows*M_B.cols*sizeof(float))	;



    return 0;
}

